#include <iostream>
#include <hip/hip_runtime.h>

#define N 100000
__device__ double atomicAddDouble(double* address, double val){
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
__global__ void sumArrayInt(int* array)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid<N&&tid!=0){
        atomicAdd(&array[0], array[tid]);
    }
}
__global__ void sumArrayDouble(double * array, unsigned long long int* result,double * res)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid<N){
//        atomicAdd(result, __double_as_longlong(array[tid]));
//        *res = __longlong_as_double(*result);
        atomicAddDouble(res,array[tid]);
    }

}


int main()
{
    int* h_array = new int[N];
    int size = N * sizeof(int);

    int* d_array;
    hipMalloc((void**)&d_array, size);

    for (int i = 0; i < N; i++)
    {
        h_array[i] = 1;
    }

    hipMemcpy(d_array, h_array, size, hipMemcpyHostToDevice);

    int numThreadsPerBlock = 1024;
//определяем необходимое количество блоков
    int numBlocks = (N+numThreadsPerBlock-1)/numThreadsPerBlock;

    sumArrayInt<<<numBlocks,numThreadsPerBlock>>>(d_array);
    hipMemcpy(h_array, d_array, size, hipMemcpyDeviceToHost);


    double *h_arrayD = new double [N];
    int sizeD = N * sizeof(double );

    double *d_arrayD;
    hipMalloc((void **) &d_arrayD, sizeD);

    for (int i = 0; i < N; i++) {
        h_arrayD[i] = 1.0;
    }
    unsigned long long h_result = 0;
    unsigned long long *d_result;
    hipMalloc((void **) &d_result, sizeof(unsigned long long));
    hipMemcpy(d_result, &h_result, sizeof(unsigned long long), hipMemcpyHostToDevice);

    double h_res = 0.0;
    double *d_res;
    hipMalloc((void **) &d_res, sizeof(double ));
    hipMemcpy(d_res, &h_res, sizeof(double ), hipMemcpyHostToDevice);

    hipMemcpy(d_arrayD, h_arrayD, sizeD, hipMemcpyHostToDevice);

    sumArrayDouble<<<numBlocks, numThreadsPerBlock>>>(d_arrayD, d_result,d_res);

    hipMemcpy(h_arrayD, d_arrayD, sizeD, hipMemcpyDeviceToHost);
    hipMemcpy(&h_res, d_res, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    std::cout << "Сумма интового массива: " << h_array[0] << std::endl;
    std::cout << "Сумма дабл массива: ";
    printf("%f \n",h_res);

    delete[] h_array;
    hipFree(d_array);
    delete[] h_arrayD;
    hipFree(d_arrayD);
    hipFree(d_result);
    hipFree(d_res);

    return 0;
}