
#define N 1000000//количество элементов массивов
#define numThreadsPerBlock 1024 // количество нитей на один блок
//ядро, которое выполняет скалярное произведение
#include "iostream"
#include <hip/hip_runtime.h>
using namespace std;
__global__ void kernel( int *a, int *b, int *c )
{
    unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;
//массивы в разделяемой памяти
    __shared__ int tempA[numThreadsPerBlock];
    __shared__ int tempB[numThreadsPerBlock];
//копирование из глобальной в разделяемую память
if(tid<N) {
    tempA[threadIdx.x] = a[tid];
    tempB[threadIdx.x] = b[tid];
    __syncthreads(); //синхронизация нитей в одном блоке

    if (threadIdx.x == 0) {
        int sum = 0;
        for (int i = 0; i < numThreadsPerBlock; i++) {
            sum += tempA[i] * tempB[i];
        }
        atomicAdd(c, sum);
    }
} else{
    tempA[threadIdx.x] = 0;
    tempB[threadIdx.x] = 0;
}
}
__global__ void kernel2( int *a, int *b, int *c )
{
    unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid<N){
        int temp = a[threadIdx.x] * b[threadIdx.x];
        atomicAdd(c, temp);
    }
}
int main( void )
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int *a, *b, c =0;
    int *dev_a, *dev_b, *dev_c;
    int size = N * sizeof( int );
    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );
    hipMalloc( (void**)&dev_c, sizeof( int ) );
    a = (int *)malloc( size );
    b = (int *)malloc( size );
    for(int i = 0 ; i<N;i++){
        a[i] = 1;
        b[i] =1;
    }
// копируем массивы на device
    hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );
//запускаем на выполнение kernel() с 1 блоком и N нитями
//определяем необходимое количество блоков
    int numBlocks = (N+numThreadsPerBlock-1)/numThreadsPerBlock;
//подставляем переменные numBlocks и numThreadsPerBlock в ядро
    hipEventRecord(start);
    kernel<<<numBlocks,numThreadsPerBlock>>>( dev_a, dev_b, dev_c );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);
//при необходимости можно добавить синхронизацию cudaDeviceSynchronize(), для ожидания
   // завершения работы на device
//копируем результат работы device на host
    hipMemcpy( &c, dev_c, sizeof( int ) , hipMemcpyDeviceToHost );
    cout<<"1 задание \nСкалярное произведение: "<<c<<endl;
    cout<<"Затраченное время на 1 000 000 элементов: "<< milliseconds<<" ms"<<endl;
    c =0;
    hipMemcpy( dev_c, &c, sizeof(int), hipMemcpyHostToDevice );
    hipEventRecord(start);
    kernel2<<<numBlocks,numThreadsPerBlock>>>( dev_a, dev_b, dev_c );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy( &c, dev_c, sizeof( int ) , hipMemcpyDeviceToHost );
    cout<<"2 задание \nСкалярное произведение: "<<c<<endl;
    cout<<"Затраченное время на 1 000 000 элементов: "<< milliseconds<<" ms"<<endl;
    free( a );
    free( b );
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
    return 0;
}