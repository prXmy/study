#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#define ARRAY_SIZE 10

__global__ void findMinValue(int* array, int* result) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < ARRAY_SIZE) {
        atomicMin(result, array[tid]);
    }
}
__global__ void findMaxValue(int* array, int* result) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < ARRAY_SIZE) {
        atomicMax(result, array[tid]);
    }
}

int main() {
    srand(time(0));
    int h_array[ARRAY_SIZE];
    int* d_array, *d_min, *d_max;
    int h_min = INT_MAX;
    int h_max = INT_MIN;
    // Заполнение массива случайными числами
    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_array[i] = rand() % 100; // Заполнение числами от 0 до 99
        std::cout << h_array[i] << " ";
    }
    std::cout << std::endl;

    // Выделение памяти на GPU
    hipMalloc(&d_array, ARRAY_SIZE * sizeof(int));
    hipMalloc(&d_min, sizeof(int));
    hipMalloc(&d_max, sizeof(int));
    // Копирование массива на GPU
    hipMemcpy(d_array, h_array, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);

    // Установим начальное значение для минимума
    hipMemcpy(d_min, &h_min, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_max, &h_max, sizeof(int), hipMemcpyHostToDevice);
    // Вычисление минимального значения через atomicMin
    int numThreadsPerBlock = 1024;
//определяем необходимое количество блоков
    int numBlocks = (ARRAY_SIZE+numThreadsPerBlock-1)/numThreadsPerBlock;
//подставляем переменные numBlocks и numThreadsPerBlock в ядро
    findMinValue<<<numBlocks,numThreadsPerBlock>>>(d_array, d_min);
    findMaxValue<<<numBlocks,numThreadsPerBlock>>>(d_array, d_max);
    // Копирование результата на хост
    hipMemcpy(&h_min, d_min, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_max, d_max, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "Минимальное значение: " << h_min << std::endl;
    std::cout << "Максимальное значение: " << h_max << std::endl;
    // Освобождение памяти на GPU
    hipFree(d_array);
    hipFree(d_min);
    hipFree(d_max);
    return 0;
}